#include "hip/hip_runtime.h"
#ifndef _FVR_CU_INTERFACE_CU_
#define _FVR_CU_INTERFACE_CU_

// Includes, System
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// Includes, Project
#ifndef _WIN32
	//#include <shrUtils.h>
#endif

#include <cutil_inline.h>

// #include "getTimeOfDay.h"

int mLoopGPU = 10; 
int gpuFilterSize = 5;

// Slice Texture
texture <float2, 2, hipReadModeElementType> inSliceTex;


// timeval gpustartTimer_, gpustopTimer_;
double gpu_time; 

// Input Slice Texture 
/* 
	Type 		: float 
	No. Components	: 2 Components After Modefying GLEW  
	Dimensions 	: 2D Texture 	 
*/  

float* rowIndex; 
float* mFilter;
size_t mPitch; 


__global__
void CreateFilter(float* fFilter, size_t fPitch, int N)
{
	// PI 
	float PI = 3.1416; 
	
	// Indecies (x,y)
	int i = threadIdx.x;
	int j = threadIdx.y; 
	
	// Filter Function 
	fFilter[(i * N) + j]  = (0.42) - (0.5 * cosf (2 * PI * (i) / (N-1))) + (0.08 * cosf (4 * PI * (i) / (N-1))); 
	fFilter[(i * N) + j] = fFilter[(i * N) + j] * (0.42 - 0.5 * cosf (2 * PI * (j) / (N-1)) + 0.08 * cosf (4 * PI * (j) / (N-1))); 
}

extern "C"
void CreateFilterGPU(dim3 grid, dim3 block, float* fFilter, size_t fPitch, int N, float* HostFilterCopied)
{
	// Filter 
	mFilter = fFilter; 
	
	// Pitch  
	mPitch = fPitch; 

	// Create Filter 
	CreateFilter <<< grid, block >>> (mFilter, fPitch, gpuFilterSize); 
	
	// Copy Back Filter Values to Check 
	hipMemcpy(HostFilterCopied, mFilter, (sizeof(float) * gpuFilterSize * gpuFilterSize), hipMemcpyDeviceToHost); 
}



// Get Pixel from 2D Slice Image, With Clamping to Slice Border
__device__ float2 GetPixel(int fX, int fY)
{
    // Only 2 Components (Real + Imaginary)
    float2 TexelValue = tex2D(inSliceTex, fX, fY);
    return TexelValue; 
}

__global__
void ProcessSliceOnCUDA(float2* fOutputSlice, int fSliceWidth, int fSliceHeight, int nSize, float* fFilter, size_t fPitch)
{
	float2 fTexValue;

	// Initialize Values
	float2 varTemp;
	varTemp.x = 0; 
	varTemp.y = 0;
	
	// Thread Index = Thread(x,y)
	int xThreadID = threadIdx.x;
	int yThreadID = threadIdx.y;
	
	// Block Index = Block(x,y)
	int fBlkWidth = blockDim.x;
	int fBlkHeight = blockDim.y;
	
	// Indecies (x,y)
	int x = blockIdx.x * fBlkWidth + xThreadID;
	int y = blockIdx.y * fBlkHeight + yThreadID;
	
	// One Side 
	int Half =  (nSize - 1) /2;
	
	/*
	// Filter Loop
	for (int u = -Half; u <= Half; u++)
	{	
		for (int v = -Half; v <= Half; v++)
		{
			if ((x + u) < 0 || (x + u) > 255 || (y + v) < 0 || (y + v) > 255)
			{
				varTemp = GetPixel(x,y); 
			}
			else 
			{		
				fTexValue = GetPixel(x + u, y + v);
				
				varTemp.x += (fTexValue.x * fFilter[((u + Half) * nSize) + (v + Half)]); 
				varTemp.y += (fTexValue.y * fFilter[((u + Half) * nSize) + (v + Half)]);
			}
		}
	}
	*/ 
	fTexValue = GetPixel(x, y);
	fOutputSlice[(y * fSliceWidth) + x].x = fTexValue.x; 
	fOutputSlice[(y * fSliceWidth) + x].y = fTexValue.y; 
	
	// Wait for Loads to Complete
	__syncthreads();
}
		   
extern "C"		   
void LaunchSliceProcessingOnCUDA
(dim3 grid, dim3 block, hipArray* fInputSlice, float2* fOutputSlice, int fSliceWidth, int fSliceHeight)
{
	// Bind Input Slice Texture To CUDA Array (At Which Processing Shall Be Done) 
	cutilSafeCall(hipBindTextureToArray(inSliceTex, fInputSlice));
	
	// Channel Format 
	struct hipChannelFormatDesc fDesc; 
	cutilSafeCall(hipGetChannelDesc(&fDesc, fInputSlice));
     	
	// gettimeofday(&gpustartTimer_, NULL);
	
	for (int i = 0; i < mLoopGPU; i++)
	{
		// Run CUDA Kernel 
		ProcessSliceOnCUDA <<< grid, block >>> (fOutputSlice, fSliceWidth, fSliceHeight, gpuFilterSize, mFilter, mPitch); 
	}

	// gettimeofday(&gpustopTimer_, NULL);
	// gpu_time = ((gpustopTimer_.tv_usec - gpustartTimer_.tv_usec) / 1000); 

	// printf("gpu time for %d is %0.15f mSec \n", mLoopGPU, gpu_time); 
	// printf("gpu normalized time for resampling process is %f mSec \n", gpu_time/mLoopGPU);
}

#endif // _FVR_CU_INTERFACE_CU_

 
