#include "hip/hip_runtime.h"

/********************************************************************
*  sample.cu
*  This is a example of the CUDA program.
*********************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

/************************************************************************/
/* Init CUDA                                                            */
/************************************************************************/
#if __DEVICE_EMULATION__

bool InitCUDA(void){return true;}

#else
bool InitCUDA(void)
{
	int count = 0;
	int i = 0;

	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}
	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA.\n");
		return false;
	}
	hipSetDevice(i);

	printf("CUDA initialized.\n");
	return true;
}

#endif
/************************************************************************/
/* Example                                                              */
/************************************************************************/
__global__ static void HelloCUDA(char* result, int num)
{
	int i = 0;
	char p_HelloCUDA[] = "Hello CUDA!";
	for(i = 0; i < num; i++) {
		result[i] = p_HelloCUDA[i];
	}
}

/************************************************************************/
/* HelloCUDA                                                            */
/************************************************************************/
int sampleCUDATest(int argc, char* argv[])
{

	if(!InitCUDA()) {
		return 0;
	}

	char	*device_result	= 0;
	char	host_result[12]	={0};

	CUDA_SAFE_CALL( hipMalloc((void**) &device_result, sizeof(char) * 11));

	unsigned int timer = 0;
	CUT_SAFE_CALL( cutCreateTimer( &timer));
	CUT_SAFE_CALL( cutStartTimer( timer));

	HelloCUDA<<<1, 1, 0>>>(device_result, 11);
	CUT_CHECK_ERROR("Kernel execution failed\n");

	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	CUT_SAFE_CALL( cutStopTimer( timer));
	printf("Processing time: %f (ms)\n", cutGetTimerValue( timer));
	CUT_SAFE_CALL( cutDeleteTimer( timer));

	CUDA_SAFE_CALL( hipMemcpy(&host_result, device_result, sizeof(char) * 11, hipMemcpyDeviceToHost));
	printf("%s\n", host_result);

	CUDA_SAFE_CALL( hipFree(device_result));
	CUT_EXIT(argc, argv);

	return 0;
}
